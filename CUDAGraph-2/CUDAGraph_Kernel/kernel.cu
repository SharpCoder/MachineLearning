#define _SIZE_T_DEFINED
#ifndef __CUDACC__
#define __CUDACC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>

extern "C" {

	// Takes 2 variables,
	// int* topology should point to an integer array.
	// float* out will be mapped to a huge output array.
	// int N is the length of the topology array.
	__global__ void kernel(int* topology, float* membank, float* weights, int TOPOLOGY_WIDTH, int NODE_WIDTH)
	{
		// Get the thread index.
		int threadIndex = threadIdx.x + (blockDim.x * blockIdx.x);
		
		// Calculate the layer.
		int layer = ( threadIndex / NODE_WIDTH );
	
		// Calculate the offset.
		int node = ( threadIndex % NODE_WIDTH );

		// Validate the datas.
		if ( layer > 0 && layer < TOPOLOGY_WIDTH + 1 ) {
			if ( node < topology[layer]) {
				
				// This is a valid case.
				// So first we need to start the loop.
				int terminate = 1000;
				float nodeOut = 0;
				bool stop = false;
				while ( !stop && terminate-- > 0) {

					// Set stop to true so that only a failure will make us iterate again.
					stop = true;

					// Now we iterate over each node above us.
					int max = layer - 1;
					for ( int i = 0; i < topology[max]; i++ ) {
						int arrayIndex = (max) * NODE_WIDTH + i;
						// Check the respsective sources.
						if ( membank[arrayIndex] == 0 ) {
							// If something hasn't been pushed to it yet, let's abort.
							stop = false;
							break;
						} else {
							// Otherwise, there is a value here! So let's add it to our collective.
							nodeOut += membank[arrayIndex] * weights[arrayIndex];
						}
					}

					if ( !stop ) continue;
					
					// Compute sigmoid.
					//nodeOut = nodeOut;//1.0 / ( 1.0 + exp(-nodeOut));

					// If we don't want to stop, it means we've added all of the nodes
					// we needed to. So let's push our value.
					membank[(layer) * NODE_WIDTH + node] = nodeOut;
				}

			}
		}

		__syncthreads();
		__shared__ float total;
		total = 0;
		for ( int i = 0; i < topology[TOPOLOGY_WIDTH - 1]; i++ ) {
			int arrayIndex = ( ( TOPOLOGY_WIDTH - 1 ) * NODE_WIDTH ) + i;
			total = membank[arrayIndex] * weights[arrayIndex];
		}

		
		__syncthreads();
		membank[0] = total;

	}

	int main()
	{
		return 0;
	}
}